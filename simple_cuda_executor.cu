#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <iostream>

template<class Function, class T, class SharedFactory>
__global__ void kernel(Function f, T* ptr_to_result, SharedFactory shared_factory)
{
  using shared_type = typename std::result_of<SharedFactory()>::type;

  extern __shared__ shared_type ptr_to_shared[];
  
  // wait for all agents to reach this point before calling the shared factory
  __syncthreads();

  // the first thread invokes the shared factory
  if(threadIdx.x == 0)
  {
    // copy construct the shared parameter from the factory
    ::new (ptr_to_shared) shared_type(shared_factory());
  }

  // wait for all agents to reach this point before invoking f
  __syncthreads();

  // all threads invoke f
  f(threadIdx.x, *ptr_to_result, *ptr_to_shared);

  // wait for all agents to finish with f's invocation before destroying the shared parameter
  __syncthreads();

  // the first thread destroys the shared parameter
  if(threadIdx.x == 0)
  {
    ptr_to_shared->~shared_type();
  }
}

// a cuda_thread_block_executor's execution function creates a group of execution agents
// which execute on a single CUDA thread block
struct cuda_thread_block_executor 
{
  // using execution_category = concurrent_execution_tag;

  template<class Function, class ResultFactory, class SharedFactory>
  typename std::result_of<ResultFactory()>::type
  bulk_sync_execute(Function f, size_t n, ResultFactory result_factory, SharedFactory shared_factory) const
  {
    // allocate storage for the result
    using result_type = typename std::result_of<ResultFactory()>::type;
    result_type* ptr_to_result = nullptr;
    hipMallocManaged(&ptr_to_result, sizeof(result_type), hipMemAttachGlobal);

    // invoke result factory and store it
    *ptr_to_result = result_factory();

    using shared_type = typename std::result_of<SharedFactory()>::type;

    // launch a kernel to create a group of execution agents
    kernel<<<1, n, sizeof(shared_type)>>>(f, ptr_to_result, shared_factory);

    // synchronize to wait for the execution agents to finish
    hipDeviceSynchronize();

    // copy the result back to the host
    result_type result = *ptr_to_result;

    // deallocate the temporary result's storage
    hipFree(ptr_to_result);

    return std::move(result);
  }
};

int main()
{
  cuda_thread_block_executor my_executor;

  auto result = my_executor.bulk_sync_execute([] __host__ __device__ (int idx, int& result, int& shared)
  {
    printf("Hello world, from agent %d\n", idx);

    // have the 8th thread add the shared parameter to the result
    if(idx == 8)
    {
      result += shared;
    }
  },
  16,
  [] __host__ __device__ () { return 7; }, // result factory
  [] __host__ __device__ () { return 13; } // shared factory
  );

  assert(result == 20);

  std::cout << "OK" << std::endl;

  return 0;
}

